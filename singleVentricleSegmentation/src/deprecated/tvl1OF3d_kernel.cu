#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <math.h> 

#include <iostream>


#include "stdio.h"

// for debugging
// #define CUDA_ERROR_CHECK
// #define CUDA_TIMING

#define cudaSafeCall( err ) __cnnCudaSafeCall( err, __FILE__, __LINE__ )

inline void __cnnCudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
  if ( hipSuccess != err )
  {
    fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line, hipGetErrorString( err ) );
    exit( -1 );
  }
#endif
  return;
}


#ifdef CUDA_TIMING
class CudaTimer
{
public:
  CudaTimer() 
  {
    hipEventCreate(&start_);
    hipEventCreate(&stop_);
  }

  ~CudaTimer() 
  {
    hipEventDestroy(start_);
    hipEventDestroy(stop_);
  }

  void start() 
  {
    hipEventRecord(start_, 0);
  }

  float elapsed() 
  {
    hipEventRecord(stop_);
    hipEventSynchronize(stop_);
    float t = 0;
    hipEventElapsedTime(&t, start_, stop_);
    return t;
  }

private:
  hipEvent_t start_;
  hipEvent_t stop_;
};
#endif


// helpers
__forceinline__ __device__ float myabs(const float x)
{
  return fabsf(x);
}

__forceinline__ __device__ double myabs(const double x)
{
  return fabs(x);
}

// CUDA kernels
// template <typename T>
// __global__ void cuda_primal_update_step_kernel(
//   torch::PackedTensorAccessor32<T,3,torch::RestrictPtrTraits> u,
//   torch::PackedTensorAccessor32<T,4,torch::RestrictPtrTraits> p,
//   torch::PackedTensorAccessor32<T,3,torch::RestrictPtrTraits> ATq,
//   const float tau, //torch::PackedTensorAccessor32<T,3,torch::RestrictPtrTraits> tau, //
//   const float hz,
//   const int Z,
//   const int M,
//   const int N,
//   const int D) 
// {
//   int ix = blockDim.x * blockIdx.x + threadIdx.x;
//   int iy = blockDim.y * blockIdx.y + threadIdx.y;
//   int iz = blockDim.z * blockIdx.z + threadIdx.z;

//   T temp = 0;
//   if (ix < N && iy < M && iz < Z)
//   {
//     // finite differences backward
//     temp += (ix > 0) ? (ix < N - 1) ? p[iz][iy][ix-1][0] - p[iz][iy][ix][0] : p[iz][iy][ix-1][0] : -p[iz][iy][ix][0];
//     temp += (iy > 0) ? (iy < M - 1) ? p[iz][iy-1][ix][1] - p[iz][iy][ix][1] : p[iz][iy-1][ix][1] : -p[iz][iy][ix][1]; 
//     temp += (iz > 0) ? (iz < Z - 1) ? (p[iz-1][iy][ix][2] - p[iz][iy][ix][2])/hz : p[iz-1][iy][ix][2]/hz : -p[iz][iy][ix][2]/hz;

//     u[iz][iy][ix] -= tau*(temp + ATq[iz][iy][ix]);     
//     //u[iz][iy][ix] -= tau[iz][iy][ix]*(temp + ATq[iz][iy][ix]);     
//   }
// }

// template <typename T>
// __global__ void cuda_dual_update_step_kernel(
//   torch::PackedTensorAccessor32<T,4,torch::RestrictPtrTraits> p,
//   torch::PackedTensorAccessor32<T,3,torch::RestrictPtrTraits> u,
//   torch::PackedTensorAccessor32<T,1,torch::RestrictPtrTraits> sigma,
//   const float hz,
//   const float lamda,
//   const int Z,
//   const int M,
//   const int N,
//   const int D) 
// {
//   int ix = blockDim.x * blockIdx.x + threadIdx.x;
//   int iy = blockDim.y * blockIdx.y + threadIdx.y;
//   int iz = blockDim.z * blockIdx.z + threadIdx.z;

//   if (ix < N && iy < M && iz < Z)
//   {
//       const int xp = ix + (ix < N - 1);
//       const int yp = iy + (iy < M - 1);
//       const int zp = iz + (iz < Z - 1);

//       const T ptmp_x = p[iz][iy][ix][0] + sigma[0]*(u[iz][iy][xp] - u[iz][iy][ix]);
//       const T ptmp_y = p[iz][iy][ix][1] + sigma[1]*(u[iz][yp][ix] - u[iz][iy][ix]);
//       const T ptmp_z = p[iz][iy][ix][2] + sigma[2]*(u[zp][iy][ix] - u[iz][iy][ix])/hz;

//       // calculate 2-norm along first dimension
//       const T denom_temp = max(sqrtf(ptmp_x*ptmp_x + ptmp_y*ptmp_y + ptmp_z*ptmp_z)/lamda, 1.);

//       // prox operation
//       p[iz][iy][ix][0] = ptmp_x / denom_temp;
//       p[iz][iy][ix][1] = ptmp_y / denom_temp;
//       p[iz][iy][ix][2] = ptmp_z / denom_temp;
//   }
// }

// template <typename T>
// __global__ void cuda_prox_l2_forward_kernel(
//   torch::PackedTensorAccessor32<T,3,torch::RestrictPtrTraits> q,
//   torch::PackedTensorAccessor32<T,1,torch::RestrictPtrTraits> sigma,
//   const int Z,
//   const int M,
//   const int N) 
// {
//   int ix = blockDim.x * blockIdx.x + threadIdx.x;
//   int iy = blockDim.y * blockIdx.y + threadIdx.y;
//   int iz = blockDim.z * blockIdx.z + threadIdx.z;

//   if (ix < N && iy < M && iz < Z)
//   {
//     // l2 prox on q 
//     q[iz][iy][ix] /= (sigma[iz] + 1.);
//   }
// }




template <typename T>
__global__ void cuda_nabla1d_fd_forward_kernel(
  const torch::PackedTensorAccessor32<T,1,torch::RestrictPtrTraits> b,
  const int NX,
  torch::PackedTensorAccessor32<T,2,torch::RestrictPtrTraits> Db)
{
  int ix = blockDim.x * blockIdx.x + threadIdx.x;

  if (ix < NX )
  {
      Db[ix][0] = (ix < NX-1) ? (b[ix+1] - b[ix]) : 0;
  }
  
}


template <typename T>
__global__ void cuda_divergence1d_fd_backward_kernel(
  const torch::PackedTensorAccessor32<T,2,torch::RestrictPtrTraits> p,
  const int NX,
  torch::PackedTensorAccessor32<T,1,torch::RestrictPtrTraits> divp)
{
  int ix = blockDim.x * blockIdx.x + threadIdx.x;

  if (ix < NX )
  {
      divp[ix] = (ix > 0) ? 
                            (ix < NX - 1 ) ? 
                               (p[ix-1][0] - p[ix][0]) 
                               : 
                               p[ix-1][0]
                              :
                              -p[ix][0];
  }
  
}


template <typename T>
__global__ void cuda_nabla2d_fd_forward_kernel(
  const torch::PackedTensorAccessor32<T,2,torch::RestrictPtrTraits> b,
  const int NY,
  const int NX,
  torch::PackedTensorAccessor32<T,3,torch::RestrictPtrTraits> Db)
{
  int ix = blockDim.x * blockIdx.x + threadIdx.x;
  int iy = blockDim.y * blockIdx.y + threadIdx.y;

  if (ix < NX && iy < NY )
  {
      Db[iy][ix][0] = (ix < NX-1) ? (b[iy][ix+1] - b[iy][ix]) : 0;
      Db[iy][ix][1] = (iy < NY-1) ? (b[iy+1][ix] - b[iy][ix]) : 0;
  }
  
}


template <typename T>
__global__ void cuda_divergence2d_fd_backward_kernel(
  const torch::PackedTensorAccessor32<T,3,torch::RestrictPtrTraits> p,
  const int NY,
  const int NX,
  torch::PackedTensorAccessor32<T,2,torch::RestrictPtrTraits> divp)
{
  int ix = blockDim.x * blockIdx.x + threadIdx.x;
  int iy = blockDim.y * blockIdx.y + threadIdx.y;

  if (ix < NX && iy < NY)
  {
      T divp_x = (ix > 0) ? 
                            (ix < NX - 1 ) ? 
                               (p[iy][ix-1][0] - p[iy][ix][0]) 
                               : 
                               p[iy][ix-1][0]
                              :
                              -p[iy][ix][0];

      T divp_y = (iy > 0) ? 
                            (iy < NY - 1 ) ? 
                               (p[iy-1][ix][1] - p[iy][ix][1]) 
                               : 
                               p[iy-1][ix][1]
                              :
                              -p[iy][ix][1];
                        
      divp[iy][ix] = divp_x + divp_y;
  }
  
}


template <typename T>
__global__ void cuda_nabla3d_fd_forward_kernel(
  const torch::PackedTensorAccessor32<T,3,torch::RestrictPtrTraits> b,
  const int NZ,
  const int NY,
  const int NX,
  const float hZ,
  torch::PackedTensorAccessor32<T,4,torch::RestrictPtrTraits> Db)
{
  int ix = blockDim.x * blockIdx.x + threadIdx.x;
  int iy = blockDim.y * blockIdx.y + threadIdx.y;
  int iz = blockDim.z * blockIdx.z + threadIdx.z;

  if (ix < NX && iy < NY && iz < NZ)
  {
      Db[iz][iy][ix][0] = (ix < NX-1) ? (b[iz][iy][ix+1] - b[iz][iy][ix]) : 0;
      Db[iz][iy][ix][1] = (iy < NY-1) ? (b[iz][iy+1][ix] - b[iz][iy][ix]) : 0;
      Db[iz][iy][ix][2] = (iz < NZ-1) ? (b[iz+1][iy][ix] - b[iz][iy][ix])/hZ : 0;
  }
  
}


template <typename T>
__global__ void cuda_divergence3d_fd_backward_kernel(
  const torch::PackedTensorAccessor32<T,4,torch::RestrictPtrTraits> p,
  const int NZ,
  const int NY,
  const int NX,
  const float hZ,
  torch::PackedTensorAccessor32<T,3,torch::RestrictPtrTraits> divp)
{
  int ix = blockDim.x * blockIdx.x + threadIdx.x;
  int iy = blockDim.y * blockIdx.y + threadIdx.y;
  int iz = blockDim.z * blockIdx.z + threadIdx.z;

  if (ix < NX && iy < NY && iz < NZ)
  {
      T divp_x = (ix > 0) ? 
                            (ix < NX - 1 ) ? 
                               (p[iz][iy][ix-1][0] - p[iz][iy][ix][0]) 
                               : 
                               p[iz][iy][ix-1][0]
                              :
                              -p[iz][iy][ix][0];

      T divp_y = (iy > 0) ? 
                            (iy < NY - 1 ) ? 
                               (p[iz][iy-1][ix][1] - p[iz][iy][ix][1]) 
                               : 
                               p[iz][iy-1][ix][1]
                              :
                              -p[iz][iy][ix][1];
                        
      T divp_z = (iz > 0) ? 
                            (iz < NZ - 1 ) ? 
                               (p[iz-1][iy][ix][2] - p[iz][iy][ix][2]) 
                               : 
                               p[iz-1][iy][ix][2]
                              :
                              -p[iz][iy][ix][2];
                        
      divp[iz][iy][ix] = divp_x + divp_y + divp_z/hZ;
  }
  
}



template <typename T>
__global__ void cuda_nabla1d_cd_forward_kernel(
  const torch::PackedTensorAccessor32<T,1,torch::RestrictPtrTraits> b,
  const int NX,
  torch::PackedTensorAccessor32<T,2,torch::RestrictPtrTraits> Db)
{
  int ix = blockDim.x * blockIdx.x + threadIdx.x;

  if (ix < NX )
  {
      Db[ix][0] = (ix > 0) ? 
                            (ix < NX-1) ? 
                              0.5*(b[ix+1] - b[ix-1]) 
                              : 
                              0.5*(b[ix] - b[ix-1]) 
                            : 
                            0.5*(b[ix+1]-b[ix]);
  }
  
}

template <typename T>
__global__ void cuda_divergence1d_cd_backward_kernel(
  const torch::PackedTensorAccessor32<T,2,torch::RestrictPtrTraits> p,
  const int NX,
  torch::PackedTensorAccessor32<T,1,torch::RestrictPtrTraits> divp)
{
  int ix = blockDim.x * blockIdx.x + threadIdx.x;

  if (ix < NX )
  {
      divp[ix] = (ix > 0) ? 
                            (ix < NX - 1 ) ? 
                               0.5*(p[ix-1][0] - p[ix+1][0]) 
                               : 
                               0.5*(p[ix-1][0] + p[ix][0]) 
                              :
                              0.5*(-p[ix][0] - p[ix+1][0]);
  }
  
}

template <typename T>
__global__ void cuda_nabla2d_cd_forward_kernel(
  const torch::PackedTensorAccessor32<T,2,torch::RestrictPtrTraits> b,
  const int NY,
  const int NX,
  torch::PackedTensorAccessor32<T,3,torch::RestrictPtrTraits> Db)
{
  int ix = blockDim.x * blockIdx.x + threadIdx.x;
  int iy = blockDim.y * blockIdx.y + threadIdx.y;

  if (ix < NX && iy < NY )
  {
      Db[iy][ix][0] = (ix > 0) ? 
                            (ix < NX-1) ? 
                              0.5*(b[iy][ix+1] - b[iy][ix-1]) 
                              : 
                              0.5*(b[iy][ix] - b[iy][ix-1]) 
                            : 
                            0.5*(b[iy][ix+1]-b[iy][ix]);

      Db[iy][ix][1] = (iy > 0) ? 
                            (iy < NY-1) ? 
                              0.5*(b[iy+1][ix] - b[iy-1][ix]) 
                              : 
                              0.5*(b[iy][ix] - b[iy-1][ix]) 
                            : 
                            0.5*(b[iy+1][ix]-b[iy][ix]);
  }
  
}


template <typename T>
__global__ void cuda_divergence2d_cd_backward_kernel(
  const torch::PackedTensorAccessor32<T,3,torch::RestrictPtrTraits> p,
  const int NY,
  const int NX,
  torch::PackedTensorAccessor32<T,2,torch::RestrictPtrTraits> divp)
{
  int ix = blockDim.x * blockIdx.x + threadIdx.x;
  int iy = blockDim.y * blockIdx.y + threadIdx.y;

  if (ix < NX && iy < NY )
  {
      T divp_x = (ix > 0) ? 
                            (ix < NX - 1 ) ? 
                               0.5*(p[iy][ix-1][0] - p[iy][ix+1][0]) 
                               : 
                               0.5*(p[iy][ix-1][0] + p[iy][ix][0]) 
                              :
                              0.5*(-p[iy][ix][0] - p[iy][ix+1][0]);

      T divp_y = (iy > 0) ? 
                            (iy < NY - 1 ) ? 
                               0.5*(p[iy-1][ix][1] - p[iy+1][ix][1]) 
                               : 
                               0.5*(p[iy-1][ix][1] + p[iy][ix][1]) 
                              :
                              0.5*(-p[iy][ix][1] - p[iy+1][ix][1]);

      divp[iy][ix] = divp_x + divp_y;
  }
  
}


template <typename T>
__global__ void cuda_nabla3d_cd_forward_kernel(
  const torch::PackedTensorAccessor32<T,3,torch::RestrictPtrTraits> b,
  const int NZ,
  const int NY,
  const int NX,
  const float hZ,
  torch::PackedTensorAccessor32<T,4,torch::RestrictPtrTraits> Db)
{
  int ix = blockDim.x * blockIdx.x + threadIdx.x;
  int iy = blockDim.y * blockIdx.y + threadIdx.y;
  int iz = blockDim.z * blockIdx.z + threadIdx.z;

  if (ix < NX && iy < NY && iz < NZ)
  {
      Db[iz][iy][ix][0] = (ix > 0) ? 
                            (ix < NX-1) ? 
                              0.5*(b[iz][iy][ix+1] - b[iz][iy][ix-1]) 
                              : 
                              0.5*(b[iz][iy][ix] - b[iz][iy][ix-1]) 
                            : 
                            0.5*(b[iz][iy][ix+1]-b[iz][iy][ix]);

      Db[iz][iy][ix][1] = (iy > 0) ? 
                            (iy < NY-1) ? 
                              0.5*(b[iz][iy+1][ix] - b[iz][iy-1][ix]) 
                              : 
                              0.5*(b[iz][iy][ix] - b[iz][iy-1][ix]) 
                            : 
                            0.5*(b[iz][iy+1][ix]-b[iz][iy][ix]);

      Db[iz][iy][ix][2] = (iz > 0) ? 
                            (iz < NZ-1) ? 
                              0.5*(b[iz+1][iy][ix] - b[iz-1][iy][ix])/hZ
                              : 
                              0.5*(b[iz][iy][ix] - b[iz-1][iy][ix])/hZ
                            : 
                            0.5*(b[iz+1][iy][ix]-b[iz][iy][ix])/hZ;
  }
  
}


template <typename T>
__global__ void cuda_divergence3d_cd_backward_kernel(
  const torch::PackedTensorAccessor32<T,4,torch::RestrictPtrTraits> p,
  const int NZ,
  const int NY,
  const int NX,
  const float hZ,
  torch::PackedTensorAccessor32<T,3,torch::RestrictPtrTraits> divp)
{
  int ix = blockDim.x * blockIdx.x + threadIdx.x;
  int iy = blockDim.y * blockIdx.y + threadIdx.y;
  int iz = blockDim.z * blockIdx.z + threadIdx.z;

  if (ix < NX && iy < NY && iz < NZ )
  {
      T divp_x = (ix > 0) ? 
                            (ix < NX - 1 ) ? 
                               0.5*(p[iz][iy][ix-1][0] - p[iz][iy][ix+1][0]) 
                               : 
                               0.5*(p[iz][iy][ix-1][0] + p[iz][iy][ix][0]) 
                              :
                              0.5*(-p[iz][iy][ix][0] - p[iz][iy][ix+1][0]);

      T divp_y = (iy > 0) ? 
                            (iy < NY - 1 ) ? 
                               0.5*(p[iz][iy-1][ix][1] - p[iz][iy+1][ix][1]) 
                               : 
                               0.5*(p[iz][iy-1][ix][1] + p[iz][iy][ix][1]) 
                              :
                              0.5*(-p[iz][iy][ix][1] - p[iz][iy+1][ix][1]);

      T divp_z = (iz > 0) ? 
                            (iz < NZ - 1 ) ? 
                               0.5*(p[iz-1][iy][ix][2] - p[iz+1][iy][ix][2]) 
                               : 
                               0.5*(p[iz-1][iy][ix][2] + p[iz][iy][ix][2]) 
                              :
                              0.5*(-p[iz][iy][ix][2] - p[iz+1][iy][ix][2]);

      divp[iz][iy][ix] = divp_x + divp_y + divp_z/hZ;
  }
  
}



// template <typename T>
// __global__ void alternative_cuda_central_differences_kernel_3D(
//   const torch::PackedTensorAccessor32<T,3,torch::RestrictPtrTraits> b,
//   const int NX,
//   const int NY,
//   const int NZ,
//   const float hZ,
//   torch::PackedTensorAccessor32<T,4,torch::RestrictPtrTraits> Db)
// {
//   int ix = blockDim.x * blockIdx.x + threadIdx.x;
//   int iy = blockDim.y * blockIdx.y + threadIdx.y;
//   int iz = blockDim.z * blockIdx.z + threadIdx.z;

//   //printf("%d \n", ix);

//   if (ix < NX && iy < NY && iz < NZ)
//   {
//       Db[ix][iy][iz][0] = (ix > 0) ? 
//                             (ix < NX-1) ? 
//                               0.5*(b[ix+1][iy][iz] - b[ix-1][iy][iz]) 
//                               : 
//                               0.5*(b[ix][iy][iz] - b[ix-1][iy][iz]) 
//                             : 
//                             0.5*(b[ix+1][iy][iz]-b[ix][iy][iz]);

//       Db[ix][iy][iz][1] = (iy > 0) ? 
//                             (iy < NY-1) ? 
//                               0.5*(b[ix][iy+1][iz] - b[ix][iy-1][iz]) 
//                               : 
//                               0.5*(b[ix][iy][iz] - b[ix][iy-1][iz]) 
//                             : 
//                             0.5*(b[ix][iy+1][iz]-b[ix][iy][iz]);

//       Db[ix][iy][iz][2] = (iz > 0) ? 
//                             (iz < NZ-1) ? 
//                               0.5*(b[ix][iy][iz+1] - b[ix][iy][iz-1])/hZ
//                               : 
//                               0.5*(b[ix][iy][iz] - b[ix][iy][iz-1])/hZ
//                             : 
//                             0.5*(b[ix][iy][iz+1]-b[ix][iy][iz])/hZ;
//   }
  
// }


// C++ kernel calls
// void cuda_primal_update_step(
//   const torch::Tensor &u,
//   const torch::Tensor &p,
//   const torch::Tensor &ATq,
//   const float tau, //const torch::Tensor &tau, //
//   const float hz)
// {
//   TORCH_CHECK(u.dim() == 3, "Expected 3d tensor for dual p");
//   TORCH_CHECK(p.dim() == 4, "Expected 4d tensor for primal u");
//   TORCH_CHECK(ATq.dim() == 3, "Expected 3d tensor for primal u");
//   //TORCH_CHECK(tau.dim() == 3, "Expected 3d tensor for stepsize tau");

//   const int Z = p.size(0);
//   const int M = p.size(1);
//   const int N = p.size(2);
//   const int D = p.size(3);
  
//   hipDeviceSynchronize();

//   const dim3 blockSize(32, 32, 1); 
//   const dim3 numBlocks((N + blockSize.x - 1) / blockSize.x,
//                       (M + blockSize.y - 1) / blockSize.y,
//                       (Z + blockSize.z - 1) / blockSize.z);

// #ifdef CUDA_TIMING
//   CudaTimer cut;
//   cut.start();
// #endif

//   AT_DISPATCH_FLOATING_TYPES(p.type(), "primal_update_step", ([&]{
//     cuda_primal_update_step_kernel<scalar_t><<<numBlocks, blockSize>>>(
//       u.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
//       p.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
//       ATq.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
//       tau, //tau.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(), //
//       hz,
//       Z,
//       M,
//       N,
//       D); 
//   }));
//   cudaSafeCall(hipGetLastError());

// #ifdef CUDA_TIMING
//   hipDeviceSynchronize();
//   std::cout << "forward time " << cut.elapsed() << std::endl;
// #endif
// }

// void cuda_dual_update_step(
//   const torch::Tensor &p,
//   const torch::Tensor &u,
//   const torch::Tensor &sigma,
//   const float hz,
//   const float lamda)
// {
//   TORCH_CHECK(p.dim() == 4, "Expected 4d tensor for dual p");
//   TORCH_CHECK(u.dim() == 3, "Expected 3d tensor for primal u");
//   TORCH_CHECK(sigma.dim() == 1, "Expected 1d tensor for stepsize sigma");

//   const int Z = p.size(0);
//   const int M = p.size(1);
//   const int N = p.size(2);
//   const int D = p.size(3);

//   const dim3 blockSize(32, 32, 1); 
//   const dim3 numBlocks((N + blockSize.x - 1) / blockSize.x,
//                       (M + blockSize.y - 1) / blockSize.y,
//                       (Z + blockSize.z - 1) / blockSize.z);

// #ifdef CUDA_TIMING
//   CudaTimer cut;
//   cut.start();
// #endif

//   AT_DISPATCH_FLOATING_TYPES(p.type(), "dual_update_step", ([&]{
//     cuda_dual_update_step_kernel<scalar_t><<<numBlocks, blockSize>>>(
//       p.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
//       u.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
//       sigma.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
//       hz,
//       lamda,
//       Z,
//       M,
//       N,
//       D); 
//   }));
//   cudaSafeCall(hipGetLastError());

// #ifdef CUDA_TIMING
//   hipDeviceSynchronize();
//   std::cout << "forward time " << cut.elapsed() << std::endl;
// #endif
// }

// void cuda_prox_l2(
//   const torch::Tensor &q,
//   const torch::Tensor &sigma)
// {
//   TORCH_CHECK(q.dim() == 3, "Expected 3d tensor");
//   TORCH_CHECK(sigma.dim() == 1, "Expected 1d tensor");

//   const int Z = q.size(0);
//   const int M = q.size(1);
//   const int N = q.size(2);

//   const dim3 blockSize(32, 32, 1); 
//   const dim3 numBlocks((N + blockSize.x - 1) / blockSize.x,
//                       (M + blockSize.y - 1) / blockSize.y,
//                       (Z + blockSize.z - 1) / blockSize.z);

// #ifdef CUDA_TIMING
//   CudaTimer cut;
//   cut.start();
// #endif

//   AT_DISPATCH_FLOATING_TYPES(q.type(), "prox_l2_forward", ([&]{
//     cuda_prox_l2_forward_kernel<scalar_t><<<numBlocks, blockSize>>>(
//       q.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
//       sigma.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
//       Z,
//       M,
//       N); 
//   }));
//   cudaSafeCall(hipGetLastError());
  
// #ifdef CUDA_TIMING
//   hipDeviceSynchronize();
//   std::cout << "forward time " << cut.elapsed() << std::endl;
// #endif
// }








torch::Tensor cuda_nabla1d_fd_forward(
  const torch::Tensor &b)
{
  TORCH_CHECK(b.dim() == 1, "Expected 1d tensor");

  const int NX = b.size(0);

  auto Db = torch::zeros({NX, 1}, b.options());

  const dim3 blockSize(512, 1, 1); 
  const dim3 numBlocks((NX + blockSize.x - 1) / blockSize.x );

#ifdef CUDA_TIMING
  CudaTimer cut;
  cut.start();
#endif

  AT_DISPATCH_FLOATING_TYPES(b.type(), "nabla1d_fd_forward", ([&]{
    cuda_nabla1d_fd_forward_kernel<scalar_t><<<numBlocks, blockSize>>>(
      b.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
      NX,
      Db.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>());
  }));
  cudaSafeCall(hipGetLastError());

#ifdef CUDA_TIMING
  hipDeviceSynchronize();
  std::cout << "forward time " << cut.elapsed() << std::endl;
#endif

  return Db;
}


torch::Tensor cuda_divergence1d_fd_backward(
  const torch::Tensor &p)
{
  TORCH_CHECK(p.dim() == 2, "Expected 2d tensor");

  const int NX = p.size(0);

  auto divp = torch::zeros({NX}, p.options());

  const dim3 blockSize(512, 1, 1); 
  const dim3 numBlocks((NX + blockSize.x - 1) / blockSize.x );

#ifdef CUDA_TIMING
  CudaTimer cut;
  cut.start();
#endif

  AT_DISPATCH_FLOATING_TYPES(p.type(), "divergence1d_fd_backward", ([&]{
    cuda_divergence1d_fd_backward_kernel<scalar_t><<<numBlocks, blockSize>>>(
      p.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
      NX,
      divp.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>());
  }));
  cudaSafeCall(hipGetLastError());

#ifdef CUDA_TIMING
  hipDeviceSynchronize();
  std::cout << "forward time " << cut.elapsed() << std::endl;
#endif

  return divp;
}


torch::Tensor cuda_nabla2d_fd_forward(
  const torch::Tensor &b)
{
  TORCH_CHECK(b.dim() == 2, "Expected 2d tensor");

  const int NY = b.size(0);
  const int NX = b.size(1);

  auto Db = torch::zeros({NY, NX, 2}, b.options());

  const dim3 blockSize(32, 32, 1); 
  const dim3 numBlocks((NX + blockSize.x - 1) / blockSize.x,
                      (NY + blockSize.y - 1) / blockSize.y );

#ifdef CUDA_TIMING
  CudaTimer cut;
  cut.start();
#endif

  AT_DISPATCH_FLOATING_TYPES(b.type(), "nabla2d_fd_forward", ([&]{
    cuda_nabla2d_fd_forward_kernel<scalar_t><<<numBlocks, blockSize>>>(
      b.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
      NY,
      NX,
      Db.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>());
  }));
  cudaSafeCall(hipGetLastError());

#ifdef CUDA_TIMING
  hipDeviceSynchronize();
  std::cout << "forward time " << cut.elapsed() << std::endl;
#endif

  return Db;
}


torch::Tensor cuda_divergence2d_fd_backward(
  const torch::Tensor &p)
{
  TORCH_CHECK(p.dim() == 3, "Expected 3d tensor");

  const int NY = p.size(0);
  const int NX = p.size(1);

  auto divp = torch::zeros({NY,NX}, p.options());

  const dim3 blockSize(32, 32, 1); 
  const dim3 numBlocks((NX + blockSize.x - 1) / blockSize.x,
                       (NY + blockSize.y - 1) / blockSize.y  );

#ifdef CUDA_TIMING
  CudaTimer cut;
  cut.start();
#endif

  AT_DISPATCH_FLOATING_TYPES(p.type(), "divergence2d_fd_backward", ([&]{
    cuda_divergence2d_fd_backward_kernel<scalar_t><<<numBlocks, blockSize>>>(
      p.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
      NY,
      NX,
      divp.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>());
  }));
  cudaSafeCall(hipGetLastError());

#ifdef CUDA_TIMING
  hipDeviceSynchronize();
  std::cout << "forward time " << cut.elapsed() << std::endl;
#endif

  return divp;
}

torch::Tensor cuda_nabla3d_fd_forward(
  const torch::Tensor &b)
{
  TORCH_CHECK(b.dim() == 3, "Expected 3d tensor");

  const int NZ = b.size(0);
  const int NY = b.size(1);
  const int NX = b.size(2);
  const float hZ = NZ / std::sqrt(0.5*NX*NX+0.5*NY*NY);

  auto Db = torch::zeros({NZ,NY,NX, 3}, b.options());

  const dim3 blockSize(16,16,3); 
  const dim3 numBlocks((NX + blockSize.x - 1) / blockSize.x,
                      (NY + blockSize.y - 1) / blockSize.y,
                      (NZ + blockSize.z - 1) / blockSize.z);

#ifdef CUDA_TIMING
  CudaTimer cut;
  cut.start();
#endif

  AT_DISPATCH_FLOATING_TYPES(b.type(), "nabla3d_fd_forward", ([&]{
    cuda_nabla3d_fd_forward_kernel<scalar_t><<<numBlocks, blockSize>>>(
      b.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
      NZ,
      NY,
      NX,
      hZ,
      Db.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>());
  }));
  cudaSafeCall(hipGetLastError());

#ifdef CUDA_TIMING
  hipDeviceSynchronize();
  std::cout << "forward time " << cut.elapsed() << std::endl;
#endif

  return Db;
}


torch::Tensor cuda_divergence3d_fd_backward(
  const torch::Tensor &p)
{
  TORCH_CHECK(p.dim() == 4, "Expected 4d tensor");

  const int NZ = p.size(0);
  const int NY = p.size(1);
  const int NX = p.size(2);
  const float hZ = NZ / std::sqrt(0.5*NX*NX+0.5*NY*NY);

  auto divp = torch::zeros({NZ,NY,NX}, p.options());

  const dim3 blockSize(16, 16, 3); 
  const dim3 numBlocks((NX + blockSize.x - 1) / blockSize.x,
                       (NY + blockSize.y - 1) / blockSize.y,
                       (NZ + blockSize.z - 1) / blockSize.z  );

#ifdef CUDA_TIMING
  CudaTimer cut;
  cut.start();
#endif

  AT_DISPATCH_FLOATING_TYPES(p.type(), "divergence3d_fd_backward", ([&]{
    cuda_divergence3d_fd_backward_kernel<scalar_t><<<numBlocks, blockSize>>>(
      p.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
      NZ,
      NY,
      NX,
      hZ,
      divp.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>());
  }));
  cudaSafeCall(hipGetLastError());

#ifdef CUDA_TIMING
  hipDeviceSynchronize();
  std::cout << "forward time " << cut.elapsed() << std::endl;
#endif

  return divp;
}





torch::Tensor cuda_nabla1d_cd_forward(
  const torch::Tensor &b)
{
  TORCH_CHECK(b.dim() == 1, "Expected 1d tensor");

  const int NX = b.size(0);

  auto Db = torch::zeros({NX, 1}, b.options());

  const dim3 blockSize(512, 1, 1); 
  const dim3 numBlocks((NX + blockSize.x - 1) / blockSize.x );

#ifdef CUDA_TIMING
  CudaTimer cut;
  cut.start();
#endif

  AT_DISPATCH_FLOATING_TYPES(b.type(), "nabla1d_cd_forward", ([&]{
    cuda_nabla1d_cd_forward_kernel<scalar_t><<<numBlocks, blockSize>>>(
      b.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
      NX,
      Db.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>());
  }));
  cudaSafeCall(hipGetLastError());

#ifdef CUDA_TIMING
  hipDeviceSynchronize();
  std::cout << "forward time " << cut.elapsed() << std::endl;
#endif

  return Db;
}

torch::Tensor cuda_divergence1d_cd_backward(
  const torch::Tensor &p)
{
  TORCH_CHECK(p.dim() == 2, "Expected 2d tensor");

  const int NX = p.size(0);

  auto divp = torch::zeros({NX}, p.options());

  const dim3 blockSize(512, 1, 1); 
  const dim3 numBlocks((NX + blockSize.x - 1) / blockSize.x );

#ifdef CUDA_TIMING
  CudaTimer cut;
  cut.start();
#endif

  AT_DISPATCH_FLOATING_TYPES(p.type(), "divergence1d_cd_backward", ([&]{
    cuda_divergence1d_cd_backward_kernel<scalar_t><<<numBlocks, blockSize>>>(
      p.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
      NX,
      divp.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>());
  }));
  cudaSafeCall(hipGetLastError());

#ifdef CUDA_TIMING
  hipDeviceSynchronize();
  std::cout << "forward time " << cut.elapsed() << std::endl;
#endif

  return divp;
}


torch::Tensor cuda_nabla2d_cd_forward(
  const torch::Tensor &b)
{
  TORCH_CHECK(b.dim() == 2, "Expected 2d tensor");

  const int NY = b.size(0);
  const int NX = b.size(1);

  auto Db = torch::zeros({NY, NX, 2}, b.options());

  const dim3 blockSize(32, 32, 1); 
  const dim3 numBlocks((NX + blockSize.x - 1) / blockSize.x,
                      (NY + blockSize.y - 1) / blockSize.y );

#ifdef CUDA_TIMING
  CudaTimer cut;
  cut.start();
#endif

  AT_DISPATCH_FLOATING_TYPES(b.type(), "nabla2d_cd_forward", ([&]{
    cuda_nabla2d_cd_forward_kernel<scalar_t><<<numBlocks, blockSize>>>(
      b.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
      NY,
      NX,
      Db.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>());
  }));
  cudaSafeCall(hipGetLastError());

#ifdef CUDA_TIMING
  hipDeviceSynchronize();
  std::cout << "forward time " << cut.elapsed() << std::endl;
#endif

  return Db;
}


torch::Tensor cuda_divergence2d_cd_backward(
  const torch::Tensor &p)
{
  TORCH_CHECK(p.dim() == 3, "Expected 3d tensor");

  const int NY = p.size(0);
  const int NX = p.size(1);

  auto divp = torch::zeros({NY,NX}, p.options());

  const dim3 blockSize(32, 32, 1); 
  const dim3 numBlocks((NX + blockSize.x - 1) / blockSize.x,
                       (NY + blockSize.y - 1) / blockSize.y );

#ifdef CUDA_TIMING
  CudaTimer cut;
  cut.start();
#endif

  AT_DISPATCH_FLOATING_TYPES(p.type(), "divergence2d_cd_backward", ([&]{
    cuda_divergence2d_cd_backward_kernel<scalar_t><<<numBlocks, blockSize>>>(
      p.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
      NY,
      NX,
      divp.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>());
  }));
  cudaSafeCall(hipGetLastError());

#ifdef CUDA_TIMING
  hipDeviceSynchronize();
  std::cout << "forward time " << cut.elapsed() << std::endl;
#endif

  return divp;
}

torch::Tensor cuda_nabla3d_cd_forward(
  const torch::Tensor &b)
{
  TORCH_CHECK(b.dim() == 3, "Expected 3d tensor");

  const int NZ = b.size(0);
  const int NY = b.size(1);
  const int NX = b.size(2);
  const float hZ = NZ / std::sqrt(0.5*NX*NX+0.5*NY*NY);

  auto Db = torch::zeros({NZ,NY,NX, 3}, b.options());

  const dim3 blockSize(16,16,3); 
  const dim3 numBlocks((NX + blockSize.x - 1) / blockSize.x,
                      (NY + blockSize.y - 1) / blockSize.y,
                      (NZ + blockSize.z - 1) / blockSize.z);

#ifdef CUDA_TIMING
  CudaTimer cut;
  cut.start();
#endif

  AT_DISPATCH_FLOATING_TYPES(b.type(), "nabla3d_cd_forward", ([&]{
    cuda_nabla3d_cd_forward_kernel<scalar_t><<<numBlocks, blockSize>>>(
      b.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
      NZ,
      NY,
      NX,
      hZ,
      Db.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>());
  }));
  cudaSafeCall(hipGetLastError());

#ifdef CUDA_TIMING
  hipDeviceSynchronize();
  std::cout << "forward time " << cut.elapsed() << std::endl;
#endif

  return Db;
}

torch::Tensor cuda_divergence3d_cd_backward(
  const torch::Tensor &p)
{
  TORCH_CHECK(p.dim() == 4, "Expected 4d tensor");

  const int NZ = p.size(0);
  const int NY = p.size(1);
  const int NX = p.size(2);
  const float hZ = NZ / std::sqrt(0.5*NX*NX+0.5*NY*NY);

  auto divp = torch::zeros({NZ,NY,NX}, p.options());

  const dim3 blockSize(16, 16, 3); 
  const dim3 numBlocks((NX + blockSize.x - 1) / blockSize.x,
                       (NY + blockSize.y - 1) / blockSize.y,
                       (NZ + blockSize.z - 1) / blockSize.z  );

#ifdef CUDA_TIMING
  CudaTimer cut;
  cut.start();
#endif

  AT_DISPATCH_FLOATING_TYPES(p.type(), "divergence3d_cd_backward", ([&]{
    cuda_divergence3d_cd_backward_kernel<scalar_t><<<numBlocks, blockSize>>>(
      p.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
      NZ,
      NY,
      NX,
      hZ,
      divp.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>());
  }));
  cudaSafeCall(hipGetLastError());

#ifdef CUDA_TIMING
  hipDeviceSynchronize();
  std::cout << "forward time " << cut.elapsed() << std::endl;
#endif

  return divp;
}


// torch::Tensor alternative_cuda_nabla3d_cd_forward(
//   const torch::Tensor &b)
// {
//   TORCH_CHECK(b.dim() == 3, "Expected 3d tensor");

//   const int NX = b.size(0);
//   const int NY = b.size(1);
//   const int NZ = b.size(2);
//   const float hZ = NZ / std::sqrt(0.5*NX*NX+0.5*NY*NY);
//   // std::cout << "in cuda_nabla3d_cd_forward:" << std::endl;
//   // std::cout << "(Nx,Ny,Nz) = (" << NX << ", " << NY << "," << NZ << ")" << std::endl;
//   // std::cout << "hZ = " << hZ << std::endl;

//   auto Db = torch::zeros({NX,NY,NZ, 3}, b.options());

//   const dim3 blockSize(16,16,3); 
//   const dim3 numBlocks((NX + blockSize.x - 1) / blockSize.x,
//                       (NY + blockSize.y - 1) / blockSize.y,
//                       (NZ + blockSize.z - 1) / blockSize.z);

// #ifdef CUDA_TIMING
//   CudaTimer cut;
//   cut.start();
// #endif

//   AT_DISPATCH_FLOATING_TYPES(b.type(), "alternative_nabla3d_cd_forward", ([&]{
//     cuda_central_differences_kernel_3D<scalar_t><<<numBlocks, blockSize>>>(
//       b.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
//       NX,
//       NY,
//       NZ,
//       hZ,
//       Db.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>());
//   }));
//   cudaSafeCall(hipGetLastError());

// #ifdef CUDA_TIMING
//   hipDeviceSynchronize();
//   std::cout << "forward time " << cut.elapsed() << std::endl;
// #endif

//   return Db;
// }